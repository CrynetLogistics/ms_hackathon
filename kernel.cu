#include "hip/hip_runtime.h"
#include <iostream>
#include "SDL.h"
#include "stdio.h"
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "structures.h"

#undef main
#define SCREEN_WIDTH 1024
#define SCREEN_HEIGHT 1000
#define DISPLAY_TIME 30000
#define THREADS_PER_BLOCK 1024
#define NUM_OF_BLOCKS 1000

#define DATA_SET_SIZE 20
#define AXIS_PRESPILL_FACTOR 3
#define UI_BOUNDS 25 //padding
#define LINE_WIDTH 4

void drawPixelRaytracer(SDL_Surface *surface , int x, int y);

int main()
{
    SDL_Window* window = NULL;
	SDL_Init(SDL_INIT_EVERYTHING);

	SDL_Surface* outSur = SDL_CreateRGBSurface(0,
                                  SCREEN_WIDTH,
                                  SCREEN_HEIGHT,
                                  32,
                                  0x00FF0000,
                                  0x0000FF00,
                                  0x000000FF,
                                  0xFF000000);


	drawPixelRaytracer(outSur, SCREEN_WIDTH, SCREEN_HEIGHT);
	//SDL_RenderPresent(renderer);
	
	SDL_SaveBMP(outSur, "hello.bmp");

	printf("done");

	//SDL_Delay(DISPLAY_TIME);
	//Destroy window
    SDL_DestroyWindow(window);
    //Quit SDL subsystems
    SDL_Quit();
    return 0;
}


__global__ void simulateIteration(colour_t* d_colourGrid){
	int index = blockIdx.x*blockDim.x+threadIdx.x;
	int x = threadIdx.x;
	int y = blockIdx.x;

	//Y-AXIS
	if(x>=UI_BOUNDS &&  x<SCREEN_WIDTH-UI_BOUNDS && y>=SCREEN_HEIGHT-AXIS_PRESPILL_FACTOR*UI_BOUNDS && y<SCREEN_HEIGHT-AXIS_PRESPILL_FACTOR*UI_BOUNDS+LINE_WIDTH){
		d_colourGrid[index].r=50;
		d_colourGrid[index].g=50;
		d_colourGrid[index].b=50;
	//X-AXIS
	}else if(y>=UI_BOUNDS &&  y<SCREEN_HEIGHT-UI_BOUNDS && x>=AXIS_PRESPILL_FACTOR*UI_BOUNDS && x<AXIS_PRESPILL_FACTOR*UI_BOUNDS+LINE_WIDTH){
		d_colourGrid[index].r=50;
		d_colourGrid[index].g=50;
		d_colourGrid[index].b=50;
	}else{
		d_colourGrid[index].r=200;
		d_colourGrid[index].g=200;
		d_colourGrid[index].b=200;
	}
}

__global__ void drawData(colour_t* d_colourGrid, int* d_dataGrid){
	int idx = threadIdx.x;
	int dataX = d_dataGrid[idx]*4;
	int dataY = d_dataGrid[idx+1];
	int colourIdx = dataX + SCREEN_WIDTH*dataY;

	d_colourGrid[colourIdx].r = 255;
	d_colourGrid[colourIdx].g = 0;
	d_colourGrid[colourIdx].b = 255;
	d_colourGrid[colourIdx+1].r = 255;
	d_colourGrid[colourIdx+1].g = 0;
	d_colourGrid[colourIdx+1].b = 255;
	d_colourGrid[colourIdx+2].r = 255;
	d_colourGrid[colourIdx+2].g = 0;
	d_colourGrid[colourIdx+2].b = 255;
	d_colourGrid[colourIdx+3].r = 255;
	d_colourGrid[colourIdx+3].g = 0;
	d_colourGrid[colourIdx+3].b = 255;
}


void drawPixelRaytracer(SDL_Surface *surface , int x, int y){
	SDL_Rect r;
	r.h = 1;
	r.w = 1;

	colour_t* h_colourGrid = (colour_t*)calloc(x*y, sizeof(colour_t));

	colour_t* d_colourGrid;
	int* d_dataGrid;

	int* h_dataGrid = (int*)malloc(sizeof(int)*DATA_SET_SIZE);
	for(int i=0; i<10; i++){
		h_dataGrid[2*i] = i;
		h_dataGrid[2*i+1] = abs(64 - (float)i*(12+i)/6);
	}

	hipMalloc((void**) &d_dataGrid, sizeof(int)*DATA_SET_SIZE);
	hipMalloc((void**) &d_colourGrid, sizeof(colour_t)*x*y);

	hipMemcpy(d_dataGrid, h_dataGrid, sizeof(int)*DATA_SET_SIZE, hipMemcpyHostToDevice);
	//hipMemcpy(d_colourGrid, h_colourGrid, sizeof(colour_t)*x*y, hipMemcpyHostToDevice);

	//calculateIntensityFromIntersections(thisLocDir, scene, col, squareSize*squareSize);
	//draws base graphics with axes
	simulateIteration<<<NUM_OF_BLOCKS,THREADS_PER_BLOCK>>>(d_colourGrid);

	//draws data
	drawData<<<1,(int)(DATA_SET_SIZE/2)>>>(d_colourGrid, d_dataGrid);

	hipMemcpy(h_colourGrid, d_colourGrid, sizeof(colour_t)*x*y, hipMemcpyDeviceToHost);

	hipFree(d_colourGrid);
	//END OF GPU CALLING CUDA CODE


	for(int i=0;i<x;i++){
		for(int j=0;j<y;j++){
			int index = j*x+i;
			r.x = i;
			r.y = j;

			if(h_colourGrid[index].r<=255 && h_colourGrid[index].g<=255 && h_colourGrid[index].b<=255){
				SDL_FillRect(surface, &r, SDL_MapRGB(surface->format, (int)h_colourGrid[index].r, (int)h_colourGrid[index].g, (int)h_colourGrid[index].b));
			}else{
				//draw bright flourescent pink for regions out of colour range nice one zl
				SDL_FillRect(surface, &r, SDL_MapRGB(surface->format, 255, 0, 255));
			}
		}
	}
	free(h_colourGrid);
	free(h_dataGrid);
}